
#include "helper/helper_c.h"
#include "helper/helper_gpu.h"
#include "CrossSpike.h"
#include "CrossSpike.cu.h"


CrossSpike::~CrossSpike()
{
	if (_proc_num > 0) {
		free_c(_recv_offset);
		free_c(_recv_start);
		free_c(_recv_num);
		free_c(_recv_data);

		free_c(_send_offset);
		free_c(_send_start);
		free_c(_send_num);
		free_c(_send_data);
	}

	if (_gpu_array) {
		gpuFree(_gpu_array->_recv_offset);
		gpuFree(_gpu_array->_recv_start);
		gpuFree(_gpu_array->_recv_num);
		gpuFree(_gpu_array->_recv_data);

		gpuFree(_gpu_array->_send_offset);
		gpuFree(_gpu_array->_send_start);
		gpuFree(_gpu_array->_send_num);
		gpuFree(_gpu_array->_send_data);

		_gpu_array->_proc_num = 0;
		_gpu_array->_min_delay = 0;
		_gpu_array->_gpu_array = NULL;

		delete _gpu_array;
	}

	_proc_num = 0;
	_min_delay = 0;
}

int CrossSpike::to_gpu()
{
	size_t size = _min_delay * _proc_num;
	size_t num_p_1 = _proc_num + 1;

	if (!_gpu_array) {
		_gpu_array = new CrossSpike;
		_gpu_array->_proc_num = _proc_num;
		_gpu_array->_min_delay = _min_delay;

		_gpu_array->_recv_offset = copyToGPU(_recv_offset, num_p_1);
		_gpu_array->_recv_start = copyToGPU(_recv_start, size+_proc_num);
		_gpu_array->_recv_num = copyToGPU(_recv_num, _proc_num);

		_gpu_array->_send_offset = copyToGPU(_send_offset, num_p_1);
		_gpu_array->_send_start = copyToGPU(_send_start, size+_proc_num);
		_gpu_array->_send_num = copyToGPU(_send_num, _proc_num);

		_gpu_array->_recv_data = copyToGPU(_recv_data, _recv_offset[_proc_num]);

		_gpu_array->_send_data = copyToGPU(_send_data, _send_offset[_proc_num]);
	} else {
		assert(_gpu_array->_proc_num == _proc_num);
		assert(_gpu_array->_min_delay == _min_delay);

		copyToGPU(_gpu_array->_recv_offset, _recv_offset, num_p_1);
		copyToGPU(_gpu_array->_recv_start, _recv_start, size+_proc_num);
		copyToGPU(_gpu_array->_recv_num, _recv_num, _proc_num);

		copyToGPU(_gpu_array->_send_offset, _send_offset, num_p_1);
		copyToGPU(_gpu_array->_send_start, _send_start, size+_proc_num);
		copyToGPU(_gpu_array->_send_num, _send_num, _proc_num);

		copyToGPU(_gpu_array->_recv_data, _recv_data, _recv_offset[_proc_num]);

		copyToGPU(_gpu_array->_send_data, _send_data, _send_offset[_proc_num]);
	}

	return 0;
}


int CrossSpike::update_gpu(const int &curr_delay, ncclComm_t &comm_gpu, hipStream_t &s)
{
	if (curr_delay > _min_delay -1) {
		if (_proc_num > _gpu_num) {
			copyFromGPU(_send_start, _gpu_array->_send_start, _proc_num * (_min_delay + 1));
			copyFromGPU(_send_data, _gpu_array->_send_data, _send_offset[_proc_num]);
		}
		msg_gpu(comm_gpu, s);
	} else {
		hipDeviceSynchronize();
		update_kernel<<<1, _proc_num>>>(_gpu_array->_send_start, _proc_num, _min_delay, curr_delay);
	}

	return 0;
}

int CrossSpike::msg_gpu(ncclComm_t &comm_gpu, hipStream_t &s)
{
	for (int i=0; i<_proc_num; i++) {
		if (i/_gpu_num == _gpu_group) {
			_send_num[i] = 0;
		} else {
			_send_num[i] = _send_start[i*(_min_delay+1)+_min_delay];
		}
	}

	// int num_size = _min_delay * _proc_num;
	// print_mpi_x32(_send_num, num_size, "Send Num");
	// print_mpi_x32(_recv_num, num_size, "To Recv Num");

	hipDeviceSynchronize();
	ncclGroupStart();
	int size = _min_delay + 1;
	int r_offset = _gpu_group * _gpu_num;
	for (int r=0; r<_gpu_num; r++) {
		if (r != _gpu_rank) {
			ncclSend(_gpu_array->_send_start + ((r_offset + r)*size), size, NCCL_INTEGER_T, r, comm_gpu, s);
			ncclRecv(_gpu_array->_recv_start + ((r_offset + r)*size), size, NCCL_INTEGER_T, r, comm_gpu, s);
		}
	}
	ncclGroupEnd();


	MPI_Alltoall(_send_start, _min_delay+1, MPI_INTEGER_T, _recv_start, _min_delay+1, MPI_INTEGER_T, MPI_COMM_WORLD);

	hipDeviceSynchronize();

	ncclGroupStart();
	for (int r=0; r<_gpu_num; r++) {
		int idx = r_offset + r;
		int num = _send_start[idx*(_min_delay+1)+_min_delay];
		if (num > 0) {
			ncclSend(_gpu_array->_send_data + _send_offset[idx], num, NCCL_NID_T, r, comm_gpu, s);
		}
		num = _recv_start[idx*(_min_delay+1)+_min_delay];
		if (num > 0) {
			ncclRecv(_gpu_array->_recv_data + _recv_offset[idx], num, NCCL_NID_T, r, comm_gpu, s);
		}
	}
	ncclGroupEnd();


	// print_mpi_x32(_recv_num, num_size, "Recv Num");

	for (int i=0; i<_proc_num; i++) {
		if (i/_gpu_num == _gpu_group) {
			_recv_num[i] = 0;
		} else {
			_recv_num[i] = _recv_start[i*(_min_delay+1)+_min_delay];
		}
	}

#ifdef ASYNC
	int ret = MPI_Ialltoallv(_send_data, _send_num, _send_offset , MPI_NID_T, _recv_data, _recv_num, _recv_offset, MPI_INTEGER_T, MPI_COMM_WORLD, &_request);
	assert(ret == MPI_SUCCESS);
#else
	int ret = MPI_Alltoallv(_send_data, _send_num, _send_offset, MPI_NID_T, _recv_data, _recv_num, _recv_offset, MPI_INTEGER_T, MPI_COMM_WORLD);
	assert(ret == MPI_SUCCESS);
#endif

	return 0;
}


int CrossSpike::fetch_gpu(const CrossMap *map, const nid_t *tables, const nsize_t *table_sizes, const nsize_t &table_cap, const int &proc_num, const int &max_delay, const int &time, const int &grid, const int &block)
{
	int delay_idx = time % (max_delay + 1);
	int curr_delay = time % _min_delay;
	fetch_kernel<<<grid, block>>>(_gpu_array->_send_data, _gpu_array->_send_offset, _gpu_array->_send_start, map->_idx2index, map->_index2ridx, tables, table_sizes, table_cap, proc_num, delay_idx, _min_delay, curr_delay);
	return 0;
}

int CrossSpike::upload_gpu(nid_t *tables, nsize_t *table_sizes, nsize_t *c_table_sizes, const nsize_t &table_cap, const int &max_delay, const int &time, const int &grid, const int &block)
{
	int curr_delay = time % _min_delay;
	if (curr_delay >= _min_delay -1) {
		copyFromGPU(c_table_sizes, table_sizes, max_delay+1);

		for (int d=0; d<_min_delay; d++) {
			int delay_idx = (time-_min_delay+2+d+max_delay)%(max_delay+1);
			for (int g=0; g<_gpu_num; g++) {
				int p = _gpu_group * _gpu_num + g;
				int start = _recv_start[p*(_min_delay+1)+d];
				int end = _recv_start[p*(_min_delay+1)+d+1];
				if (end > start) {
					gpuMemcpy(tables + table_cap * delay_idx + c_table_sizes[delay_idx], _gpu_array->_recv_data + _recv_offset[p] + start, end - start);
					c_table_sizes[delay_idx] += end - start;
				}
			}
		}

#ifdef ASYNC
		MPI_Status status_t;
		int ret = MPI_Wait(&_request, &status_t);
		assert(ret == MPI_SUCCESS);
#endif

		for (int d=0; d < _min_delay; d++) {
			int delay_idx = (time-_min_delay+2+d+max_delay)%(max_delay+1);
			for (int p = 0; p<_proc_num; p++) {
				int start = _recv_start[p*(_min_delay+1)+d];
				int end = _recv_start[p*(_min_delay+1)+d+1];
				if (end > start && (p/_gpu_num != _gpu_group)) {
					assert(c_table_sizes[delay_idx] + end - start <= table_cap);
					copyToGPU(tables + table_cap*delay_idx + c_table_sizes[delay_idx], _recv_data + _recv_offset[p] + start, end-start);
					c_table_sizes[delay_idx] += end - start;
				}
			}
		}
		copyToGPU(table_sizes, c_table_sizes, max_delay+1);

		{ // Reset
			gpuMemset(_gpu_array->_recv_start, 0, _min_delay * _proc_num + _proc_num);
			gpuMemset(_gpu_array->_send_start, 0, _min_delay * _proc_num + _proc_num);

			memset_c(_recv_num, 0, _proc_num);
			memset_c(_send_num, 0, _proc_num);
		}
	}

	return 0;
}


